#include "hip/hip_runtime.h"
//  Copyright (c) 2021 ETH Zurich
//
//  SPDX-License-Identifier: BSL-1.0
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <hpx/init.hpp>
#include <hpx/modules/async_cuda.hpp>
#include <hpx/modules/execution.hpp>
#include <hpx/modules/executors.hpp>
#include <hpx/modules/testing.hpp>

__global__ void dummy_kernel()
{
    printf("Hello from device kernel.\n");
}

struct dummy
{
    void operator()() const
    {
        std::cout << "Hello from host kernel." << std::endl;
    }

    void operator()(hipStream_t stream) const
    {
        dummy_kernel<<<1, 1, 0, stream>>>();
    }
};

int hpx_main()
{
    using namespace hpx::cuda::experimental;
    using namespace hpx::execution::experimental;

    // Pure stream executor
    {
        std::cerr << "Scheduling no transforms" << std::endl;
        sync_wait(schedule(stream_executor{hipStream_t{}}));
        std::cerr << "Waited for results" << std::endl;
    }

    {
        std::cerr << "Scheduling one transform" << std::endl;
        sync_wait(
            transform(schedule(stream_executor{hipStream_t{}}), dummy{}));
        std::cerr << "Waited for results" << std::endl;
    }

    {
        std::cerr << "Scheduling three transforms" << std::endl;
        auto s1 = transform(schedule(stream_executor{hipStream_t{}}), dummy{});
        auto s2 = transform(s1, dummy{});
        sync_wait(s2);
        std::cerr << "Waited for results" << std::endl;
    }

    // Mixing stream executor with host executor
    {
        std::cerr << "Scheduling no transforms" << std::endl;
        auto s1 = schedule(stream_executor{hipStream_t{}});
        auto s2 = on(s1, executor{});
        auto s3 = on(s2, stream_executor{hipStream_t{}});
        sync_wait(s3);
        std::cerr << "Waited for results" << std::endl;
    }

    {
        std::cerr << "Scheduling no transforms" << std::endl;
        auto s1 = schedule(executor{});
        auto s2 = on(s1, stream_executor{hipStream_t{}});
        auto s3 = on(s2, executor{});
        sync_wait(s3);
        std::cerr << "Waited for results" << std::endl;
    }

    {
        std::cerr << "Scheduling one transform on each context" << std::endl;
        auto s1 = schedule(stream_executor{hipStream_t{}});
        auto s2 = transform(s1, dummy{});
        auto s3 = on(s2, executor{});
        auto s4 = transform(s3, dummy{});
        auto s5 = on(s4, stream_executor{hipStream_t{}});
        auto s6 = transform(s5, dummy{});
        sync_wait(s6);
        std::cerr << "Waited for results" << std::endl;
    }

    {
        std::cerr << "Scheduling one transform on each context" << std::endl;
        auto s1 = schedule(executor{});
        auto s2 = transform(s1, dummy{});
        auto s3 = on(s2, stream_executor{hipStream_t{}});
        auto s4 = transform(s3, dummy{});
        auto s5 = on(s4, executor{});
        auto s6 = transform(s5, dummy{});
        sync_wait(s6);
        std::cerr << "Waited for results" << std::endl;
    }

    return hpx::finalize();
}

int main(int argc, char* argv[])
{
    HPX_TEST_EQ_MSG(
        hpx::init(argc, argv), 0, "HPX main exited with non-zero status");

    return hpx::util::report_errors();
}
